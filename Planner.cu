#include "hip/hip_runtime.h"
#include <math.h>
#include "State.h"
#include <vector>
#include <stdlib.h>
#include <stdio.h>
 
#define	STARTING_VALUE -1
#define OBSTACLE_VALUE -2
#define GOAL_VALUE -3
#define BLOCK_SIZE 256

__device__ void retrieveNeighborsLocation(StateStruct *state, Point *neighbors) {
	Point N;
	N.x = state->x; N.y = state->y-1;	*neighbors = N;
	Point S;
	S.x = state->x; S.y = state->y+1;	*(neighbors+1) = S;
	Point E;
	E.x = state->x-1; E.y = state->y;	*(neighbors+2) = E;
	Point W;
	W.x = state->x+1; W.y = state->y;	*(neighbors+3) = W;
	Point NE;
	NE.x = state->x-1; NE.y = state->y-1;	*(neighbors+4) = NE;
	Point NW;
	NW.x = state->x+1; NW.y = state->y-1;	*(neighbors+5) = NW;
	Point SE;
	SE.x = state->x-1; SE.y = state->y+1;	*(neighbors+6) = SE;
	Point SW; 
	SW.x = state->x+1; SW.y = state->y+1;	*(neighbors+7) = SW;
}

__device__ float distance(StateStruct *from, StateStruct *to) {
	if(from->x == to->x) {
		return fabs((float)(to->y - from->y));
	}
	else if(from->y == to->y) {
		return fabs((float)(to->x - from->x));
	}
	else {
		return sqrt(fabs((float)(to->x - from->x))*fabs((float)(to->x - from->x)) +
			fabs((float)(to->y - from->y))*fabs((float)(to->y - from->y)));
	}
}

__device__ StateStruct *retrieveStateAtLocation(Point location, StateStruct *texture, int rows, int columns) {
	StateStruct* ptr = &texture[location.x+columns*location.y];
	return ptr;
}

__device__ int withinBounds(Point pt, int rows, int columns) { 
	return ((pt.x >= 0 && pt.x < columns) && (pt.y >= 0 && pt.y < rows));
}

__device__ int stateNeedsUpdate(StateStruct* state) {
	return state->g == STARTING_VALUE || state->g == GOAL_VALUE;
}

__device__ int stateIsObstacle(StateStruct* state) {
	return state->costToReach > 10.0f;
}

__device__ int isGoalState(StateStruct* state) {
	return state->g == 0.0f;
}

//GPU kernel that computes the planner solution.
__global__ void computeCostsKernel(StateStruct *current_texture, StateStruct *texture_copy, int rows, int columns, int *check, int *locality, float maxCost, bool allAgentsReached) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < columns && y < rows) {
		Point pt; pt.x = x; pt.y = y;
		StateStruct *state = retrieveStateAtLocation(pt, current_texture, rows, columns);

		if (withinBounds(pt, rows, columns)) {
			if(!stateIsObstacle(state) && !isGoalState(state)) {
				//if the state is an obstacle, do not compute neighbors
				Point neighbors[8];
				retrieveNeighborsLocation(state, &neighbors[0]);

				int i;
				for (i = 0; i < 8; ++i) {
					if (withinBounds(neighbors[i], rows, columns)) {
						StateStruct *neighbor = retrieveStateAtLocation(neighbors[i], texture_copy, rows, columns);
						if (stateIsObstacle(neighbor)) //if neighbor is an obstacle, do not use it as a possible neighbor
							continue;
						float newg = neighbor->g + distance(neighbor, state) * state->costToReach;
						if ((newg < state->g || stateNeedsUpdate(state)) && !stateNeedsUpdate(neighbor)) {
							state->predx = neighbors[i].x;
							state->predy = neighbors[i].y;
							state->g = newg;
							if (*locality == 1) {
								*check = 0;
							} else if (*locality == 2) {
								if (state->g < maxCost || !allAgentsReached) {
									*check = 0;
								}
							} else if (*locality == 0 && allAgentsReached) {
								*check = 1;
							}
						}
					}
				}
				Point predPt; predPt.x = state->predx; predPt.y = state->predy;
				StateStruct *selectedPredecessorCopy = retrieveStateAtLocation(predPt, texture_copy, rows, columns);
				state->inconsistent = false;
				if ((selectedPredecessorCopy != NULL && selectedPredecessorCopy->inconsistent) || stateIsObstacle(selectedPredecessorCopy)) {
					//if predecessor from read-only is inconsistent - clear inconsistent flag in write-only and mark state as inconsistent in write-only
					current_texture[state->predy*columns+state->predx].inconsistent = false;
					state->inconsistent = true;
					state->g = STARTING_VALUE;
				} 
			}
		}
	}	
}

__global__ void checkForInconsistency(StateStruct* texture, int rows, int columns, int* flag) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < columns && y < rows) {
		StateStruct* state = &texture[y*columns+x];
		if (state->inconsistent) {
			*flag = 1;
		}
	}
}

double showMemoryUsage1()
{
	size_t free_byte;
	size_t total_byte;
	hipMemGetInfo(&free_byte, &total_byte);

	double free_db = (double)free_byte;
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;
	printf("GPU memory usage: used = %f MB", used_db/1024.0);
	return used_db;
}

/*Finds agent largest g-value. This is used to stop the planner earlier and still get an optimal solution*/ 
float agentsMaxCost(StateStruct* texture, int columns, int agentCount, StateStruct* agents) {
	float maxCost = -10000.0f;
	for (int i = 0; i < agentCount; i++)  {
		StateStruct agent = texture[columns*agents[i].y+agents[i].x];
		if (agent.g > maxCost) {
			maxCost = agent.g;
		}
	}
	return maxCost;
}

bool agentsReached(StateStruct* texture, int columns, int agentCount, StateStruct* agents) {
	for (int i = 0; i < agentCount; i++) {
		StateStruct agent = texture[columns*agents[i].y+agents[i].x];
		if (agent.g < 0.0f) {
			return false;
		}
	}
	return true;
}

/*Method that calls the kernel. The blocks and grid sizes could be improved, depending on the kind of GPU you have.
Also, there are several things that can be done in the kernel to improve performance. We can also talk about it.*/
extern "C" int computeCostsCuda(StateStruct* texture, int rows, int columns, int locality, int agentCount, StateStruct* agents, int maxIterations = 0) {
	int *locality_dev, *consistencyCheck, *consistencyCheck_dev, *flag, *flag_dev;
	
	int blockLength = sqrt((double)BLOCK_SIZE); 
	int gridLength = ceil((double)rows/(double)blockLength);
	
	dim3 blocks(gridLength, gridLength, 1);
	dim3 threads(blockLength, blockLength, 1);

	
	StateStruct *texture_device, *texture_device_copy;
	hipMalloc((void**)&texture_device, (rows*columns)*sizeof(StateStruct));
	hipMalloc((void**)&texture_device_copy, (rows*columns)*sizeof(StateStruct));
	//make a two copies of the initial map
	hipMemcpy(texture_device, texture, (rows*columns)*sizeof(StateStruct), hipMemcpyHostToDevice);
	hipMemcpy(texture_device_copy, texture, (rows*columns)*sizeof(StateStruct), hipMemcpyHostToDevice);


	hipMalloc((void**)&locality_dev, sizeof(int));
	int* locality_ptr = (int*)malloc(sizeof(int));
	*locality_ptr = locality;

	hipMalloc((void**)&consistencyCheck_dev, sizeof(int));
	consistencyCheck = (int*)malloc(sizeof(int));

	hipMalloc((void**)&flag_dev, sizeof(int));
	flag = (int*)malloc(sizeof(int));

	int iterations = 0;
	do {
		//set flag to 0 to check for changes
		if (locality == 1 || locality == 2) {
			*consistencyCheck = 1;
		} else {
			*consistencyCheck = 0;
		}
		*flag = 0;
		hipMemcpy(locality_dev, locality_ptr, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(consistencyCheck_dev, consistencyCheck, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(flag_dev, flag, sizeof(int), hipMemcpyHostToDevice);

		bool allAgentsReached = agentsReached(texture, columns, agentCount, agents);
		float maxCost;
		if (allAgentsReached) {
			maxCost = agentsMaxCost(texture, columns, agentCount, agents);
		}
		computeCostsKernel<<<blocks, threads>>>(texture_device, texture_device_copy, rows, columns, consistencyCheck_dev, locality_dev, maxCost, allAgentsReached);
		
		checkForInconsistency<<<blocks, threads>>>(texture_device, rows, columns, flag_dev);
		
		StateStruct* temp = texture_device;
		texture_device = texture_device_copy;
		texture_device_copy = temp;
		iterations++;

		
		hipMemcpy(texture, texture_device, (rows*columns)*sizeof(StateStruct), hipMemcpyDeviceToHost);
		hipMemcpy(consistencyCheck, consistencyCheck_dev, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(flag, flag_dev, sizeof(int), hipMemcpyDeviceToHost);
	} while((*consistencyCheck == 0 || *flag == 1) && iterations != maxIterations);

	showMemoryUsage1();


	hipFree(texture_device); hipFree(texture_device_copy);

	printf("Result was: %i\n\n", *consistencyCheck);
	printf("Number of iterations: %i\n\n", iterations);

	return 1;
}

__global__ void clearTextureValuesKernel(StateStruct* texture, int rows, int columns) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < columns && y < rows) {
		StateStruct* state = &texture[y*columns+x];
		if (!stateIsObstacle(state)) {
			state->g = STARTING_VALUE;
			state->predx = state->predy = -1;
		}
	}

}

extern "C" void clearTextureValues(StateStruct* texture, int rows, int columns) {
	StateStruct* texture_dev;
	int blockLength = sqrt((double)BLOCK_SIZE); 
	int gridLength = ceil((double)rows/(double)blockLength);
	dim3 blocks(gridLength, gridLength, 1);
	dim3 threads(blockLength, blockLength, 1);

	hipMalloc((void**)&texture_dev, ((rows*columns)*sizeof(StateStruct)));
	hipMemcpy(texture_dev, texture, (columns*rows)*sizeof(StateStruct), hipMemcpyHostToDevice);
	clearTextureValuesKernel<<<blocks, threads>>> (texture_dev, rows, columns);
	hipMemcpy(texture, texture_dev, (columns*rows)*sizeof(StateStruct), hipMemcpyDeviceToHost);

	hipFree(texture_dev);
}

__device__ bool equals(float a, float b)
{
	if (fabs(a - b) < 0.0001) {
		return true;
	}
	return false;
}

/*Kernel for update after obstacle movement.*/
__global__ void propagateAfterObstacleMovementKernel(StateStruct* texture, StateStruct* texture_copy, int* propagateUpdate, int rows, int columns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < columns && y < rows) {
		StateStruct state = texture_copy[y*columns+x];
		if (state.predx > -1 && state.predy > -1)
		{
			StateStruct predecessor = texture_copy[state.predy*columns+state.predx];
			float transitionCost = distance(&state, &predecessor) * state.costToReach;
			if ((stateIsObstacle(&predecessor) || stateIsObstacle(&state) || !equals(state.g, (predecessor.g + transitionCost))) && !isGoalState(&state))
			{
				texture[y*columns+x].predx = -1;
				texture[y*columns+x].predy = -1;
				texture[y*columns+x].g = STARTING_VALUE;
				
				*propagateUpdate = 1;
			}
		}
	}
}

extern "C" void propagateUpdateAfterObstacleMovement(StateStruct* texture, int rows, int columns)
{
	StateStruct* texture_dev, *texture_dev_copy;
	int blockLength = sqrt((double)BLOCK_SIZE); 
	int gridLength = ceil((double)rows/(double)blockLength);
	dim3 blocks(gridLength, gridLength, 1);
	dim3 threads(blockLength, blockLength, 1);

	hipMalloc((void**)&texture_dev, ((rows*columns)*sizeof(StateStruct)));
	hipMemcpy(texture_dev, texture, (columns*rows)*sizeof(StateStruct), hipMemcpyHostToDevice);
		
	hipMalloc((void**)&texture_dev_copy, ((rows*columns)*sizeof(StateStruct)));
	hipMemcpy(texture_dev_copy, texture, (columns*rows)*sizeof(StateStruct), hipMemcpyHostToDevice);

	int* propagateUpdate = (int*)malloc(sizeof(int));
	
	int* propagateUpdate_dev;
	hipMalloc((void**)&propagateUpdate_dev, sizeof(int));
	do {
		*propagateUpdate = 0;
		hipMemcpy(propagateUpdate_dev, propagateUpdate, sizeof(int), hipMemcpyHostToDevice);

		propagateAfterObstacleMovementKernel<<<blocks, threads>>>(texture_dev, texture_dev_copy, propagateUpdate_dev, rows, columns);
		hipMemcpy(texture_dev_copy, texture_dev, (rows*columns)*sizeof(StateStruct), hipMemcpyDeviceToDevice);

		hipMemcpy(propagateUpdate, propagateUpdate_dev, sizeof(int), hipMemcpyDeviceToHost);
	} while(*propagateUpdate == 1);

	hipMemcpy(texture, texture_dev, (columns*rows)*sizeof(StateStruct), hipMemcpyDeviceToHost);

	hipFree(texture_dev); hipFree(texture_dev_copy); hipFree(propagateUpdate_dev);
}